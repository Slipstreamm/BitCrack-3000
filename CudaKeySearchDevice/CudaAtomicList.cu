#include "hip/hip_runtime.h"
#include "CudaAtomicList.h"
#include "CudaAtomicList.cuh"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

static __constant__ void *_LIST_BUF[1];
static __constant__ unsigned int *_LIST_SIZE[1];


__device__ void atomicListAdd(void *info, unsigned int size)
{
	unsigned int count = atomicAdd(_LIST_SIZE[0], 1);

	unsigned char *ptr = (unsigned char *)(_LIST_BUF[0]) + count * size;

	memcpy(ptr, info, size);
}

static hipError_t setListPtr(void *ptr, unsigned int *numResults)
{
	hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(_LIST_BUF), &ptr, sizeof(void *));

	if(err) {
		printf("cudaatomiclist setListPtr: hipMemcpyToSymbol list_buf error!");
		return err;
	}

	err = hipMemcpyToSymbol(HIP_SYMBOL(_LIST_SIZE), &numResults, sizeof(unsigned int *));
        if (err) {
		printf("cudaatomiclist setListPtr: hipMemcpyToSymbol list_size error!");
		return err;
	}
	return err;
}


hipError_t CudaAtomicList::init(unsigned int itemSize, unsigned int maxItems)
{
	_itemSize = itemSize;

	// The number of results found in the most recent kernel run
	_countHostPtr = NULL;
	hipError_t err = hipHostAlloc(&_countHostPtr, sizeof(unsigned int), hipHostMallocMapped);
	if(err) {
		printf("cudaAtomicList::init: hipHostAlloc countHostPtr error!");
		goto end;
	}

	// Number of items in the list
	_countDevPtr = NULL;
	err = hipHostGetDevicePointer(&_countDevPtr, _countHostPtr, 0);
	if(err) {
		printf("cudaAtomicList::init: hipHostGetDevicePointer countDevicePte error!");
		goto end;
	}
	*_countHostPtr = 0;

	// Storage for results data
	_hostPtr = NULL;
	err = hipHostAlloc(&_hostPtr, itemSize * maxItems, hipHostMallocMapped);
	if(err) {
		printf("cudaAtomicList::init: hipHostAlloc hostPtr error!");
		goto end;
	}

	// Storage for results data (device to host pointer)
	_devPtr = NULL;
	err = hipHostGetDevicePointer(&_devPtr, _hostPtr, 0);

	if(err) {
		printf("cudaAtomicList::init: hipHostGetDevicePointer devPtr error!");
		goto end;
	}

	err = setListPtr(_devPtr, _countDevPtr);

end:
	if(err) {
		hipHostFree(_countHostPtr);

		hipFree(_countDevPtr);

		hipHostFree(_hostPtr);

		hipFree(_devPtr);
	}

	return err;
}

unsigned int CudaAtomicList::size()
{
	return *_countHostPtr;
}

void CudaAtomicList::clear()
{
	*_countHostPtr = 0;
}

unsigned int CudaAtomicList::read(void *ptr, unsigned int count)
{
	if(count >= *_countHostPtr) {
		count = *_countHostPtr;
	}

	memcpy(ptr, _hostPtr, count * _itemSize);

	return count;
}

void CudaAtomicList::cleanup()
{
	hipHostFree(_countHostPtr);

	hipFree(_countDevPtr);

	hipHostFree(_hostPtr);

	hipFree(_devPtr);
}
